#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// CUDA kernel for Fibonacci sequence
__global__
void fibonacciCuda(int N, unsigned long long* d_Fibonacci) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        if (idx == 0) d_Fibonacci[idx] = 0;
        else if (idx == 1) d_Fibonacci[idx] = 1;
        else d_Fibonacci[idx] = d_Fibonacci[idx - 1] + d_Fibonacci[idx - 2];
    }
}

void fibonacciCPU(int N, unsigned long long* h_Fibonacci) {
    if (N == 0) return;

    h_Fibonacci[0] = 0;

    if (N == 1) return;

    h_Fibonacci[1] = 1;

    for (int i = 2; i < N; i++) {
        h_Fibonacci[i] = h_Fibonacci[i - 1] + h_Fibonacci[i - 2];
    }
}

int main()
{
    int N = 220;

    unsigned long long* h_Fibonacci = new unsigned long long[N];
    unsigned long long* d_Fibonacci;

    size_t size = N * sizeof(unsigned long long);
    hipMalloc((void**)&d_Fibonacci, size);
   hipMemset(d_Fibonacci, 0, size);

    // Measure CPU execution
    clock_t cpuStart = clock();
    fibonacciCPU(N, h_Fibonacci);
    clock_t cpuEnd = clock();

    double cpuTime = (double)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;

    cout << "CPU execution time: " << cpuTime << " seconds" << endl;

    // Measure GPU execution
    clock_t gpuStart = clock();

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    fibonacciCuda<<<blocks, threads>>>(N, d_Fibonacci);
    hipDeviceSynchronize();

    clock_t gpuEnd = clock();

    double gpuTime = (double)(gpuEnd - gpuStart) / CLOCKS_PER_SEC;

    cout << "GPU execution time: " << gpuTime << " seconds" << endl;

    // Retrieve the results back to host
   hipMemcpy(h_Fibonacci, d_Fibonacci, size, hipMemcpyDeviceToHost);

    cout << "First 10 Fibonacci numbers:" << endl;
    for (int i = 0; i < 10; ++i) {
        cout << h_Fibonacci[i] << " "; 
    }
    cout << endl;

    delete[] h_Fibonacci;
   hipFree(d_Fibonacci);

    return 0;
}

